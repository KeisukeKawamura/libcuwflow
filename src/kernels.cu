#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2013 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */


/**
* \file kernels.cu
*  This file contains the implementation of kernels which will be used to acelerate the algorithm.
*  It contains CUDA code from CUDA SDK with some modifications.
*
* @author Fernando Cobo Aguilera
* @date 30/11/2014
*/


#include "kernels.cuh"


/**
* \fn int binarySearchInclusive(float val, float *data, int L, int stride)
* \brief Helper function of mergeSortSharedKernel kernel which finds the right position of a 
  specified input value to get an ordered vector (Bounds of the array included)
* \param val Specified input value
* \param data Array which i been used
* \param stride Level number of the merge sort (power of two)
* \return the position found  
*/
 __device__ int binarySearchInclusive(float val, float *data, int stride)
{

    int pos = 0;

    for (; stride > 0; stride >>= 1)
    {
        int newPos = stride;

        if (data[newPos - 1] <= val)
            pos = newPos;
    }

    return pos;
}

/**
* \fn  int binarySearchExclusive(float val, float *data, int stride)
* \brief Helper function of mergeSortSharedKernel kernel which finds the right position of a 
  specified input value to get an ordered vector. (Bounds of the array excluded)
* \param val Specified input value
* \param data Array which is been used
* \param stride Level number of the merge sort (power of two)
*/
 __device__ int binarySearchExclusive(float val, float *data, int stride)
{

    int pos = 0;

    for (; stride > 0; stride >>= 1)
    {
        int newPos = stride;

        if (data[newPos - 1] < val)
            pos = newPos;
    }

    return pos;
}

 /**
* \fn void mergeSortSharedKernel(float *d_SrcKey, float* d_elementNine, int sizeSharedMemory)
* \brief Bottom-level merge sort (binary search-based) kernel. It is a variante of the original
   code from CUDA kit, adapted to merge several arrays of 9 elements.
* \param d_src_key Source vector with all the 8-elements arrays to order. The size must be a multiple of 8
* \param d_element_nine Source vector with all the ninth elements of each 8-elements array from d_SrcKey
* \param size_shared_memory Size of the shared memory due to the memory is allocated dynamically

*/
__global__ void mergeSortSharedKernel(float *d_src_key, float* d_element_nine, int size_shared_memory)
{
	//Dynamic shared memory
	extern __shared__ float s_key[];

    d_src_key += blockIdx.x * size_shared_memory + threadIdx.x;
 
	//Loading array, two accesses per thread
    s_key[threadIdx.x +                      0] = d_src_key[0];
    s_key[threadIdx.x + (size_shared_memory / 2)] = d_src_key[(size_shared_memory / 2)];


	int stride;

	//Loop adapted to order arrays of 8 elements, rather than a whole array of 8*x elements
    for (stride = 1; stride < 8; stride *= 2)
    {

        int     lPos = threadIdx.x & (stride - 1);
        float *baseKey = s_key + 2 * (threadIdx.x - lPos);

        __syncthreads();
        float keyA = baseKey[lPos +      0];
        float keyB = baseKey[lPos + stride];
        int posA = binarySearchExclusive(keyA, baseKey + stride, stride) + lPos;
        int posB = binarySearchInclusive(keyB, baseKey +      0, stride) + lPos;

        __syncthreads();
        baseKey[posA] = keyA;
        baseKey[posB] = keyB;
    }
	

    __syncthreads();

	//Extra code from the original version to compare the ninth element of each array with the 3th and 4th element
	//of each 8-elements array, which are already ordered.
	float aux;
	if( threadIdx.x % 4 == 0){

		float lowerBound  = s_key[threadIdx.x*2 + 3];
		float higherBound = s_key[threadIdx.x*2 + 4];
		
		//The ninth element is considered as the medianis to say
		aux = d_element_nine[blockIdx.x*256 + (threadIdx.x / 4)];

		//Check if the 3th element is bigger than the ninth element
		if(aux <= lowerBound)
			d_element_nine[blockIdx.x*256 + (threadIdx.x / 4)] = lowerBound;
		else{
			//Check if the 4th element is smaller than the ninth element
		    if(aux >= higherBound)
			 d_element_nine[blockIdx.x*256 + (threadIdx.x / 4)] = higherBound;	
		}
	}
}

/**
* \fn void scan(float *g_odata, float *g_idata)
* \brief This kenel calculates the parallel prefix sum or scan of several arrays.
   The elements of every array are non-contiguos, that is to say, there is a stride 
* \param g_odata Output array
* \param g_idata Input array
*/
__global__ void scan(float *g_odata, float *g_idata){

	extern __shared__ float temp[]; // allocated on invocation

	int tid = threadIdx.x;

	//Loading array to shared memory with the corresponding stride
	temp[tid] = g_idata[threadIdx.x*gridDim.x + blockIdx.x + (blockIdx.y*gridDim.x*blockDim.x)];

	float aux;

	__syncthreads();

	for(int offset = 1; offset < blockDim.x; offset *= 2){

		if( tid >= offset){

			aux = temp[tid-offset];
			__syncthreads();

			temp[tid] += aux;
		}
		else{
			temp[tid] = temp[tid];
		}
	}

	//Loading results to output array
	g_odata[threadIdx.x*gridDim.x + blockIdx.x + (blockIdx.y*gridDim.x*blockDim.x)] = temp[tid]; // write output 
}


void mergeSort(float *d_src_key, hipStream_t stream, int n_blocks, int n_threads, int shared_memory,float* d_element_nine)
{
	mergeSortSharedKernel<<<n_blocks, n_threads, shared_memory*sizeof(float),stream>>>(d_src_key,d_element_nine, shared_memory);
}


void scanKernel(float *d_idata, hipStream_t stream,int width, dim3 dims, float *d_odata){

	scan<<<dims,width,sizeof(float)*width,stream>>>(d_odata, d_idata);
}
